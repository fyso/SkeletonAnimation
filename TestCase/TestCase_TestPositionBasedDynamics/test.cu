#include "hip/hip_runtime.h"
#include "pch.h"
#include "ShapeMatchingConstraint.h"
#include "PositionBasedDynamics.cu"

class CTestPositionBasedDynamics : public testing::Test
{
protected:
	virtual void TearDown() override
	{
		_freeCuda();

		delete m_pShapeMatchingConstraint;
		m_pShapeMatchingConstraint = nullptr;
	}

	void _initShapeMatchingParticles()
	{
		m_ParticleData[0].setInfo({ 0, 0, 0 }, m_Gravity);
		m_ParticleData[1].setInfo({ 0, 1.0, 0 }, m_Gravity);
		m_ParticleData[2].setInfo({ 1.0, 0, 0 }, m_Gravity);
		m_ParticleData[0].setMass(0.0);

		m_pShapeMatchingConstraint = new Animation::CShapeMatchingConstraint(m_ParticleCount);
		const std::vector<int> ParticleIndices = { 0, 1, 2 };
		const std::vector<int> ClusterCount(m_ParticleCount, 1);
		const std::vector<double> Stiffness(m_ParticleCount, 1.0);
		m_pShapeMatchingConstraint->initializeConstraint(m_ParticleData, ParticleIndices, ClusterCount, Stiffness);

		int Size = m_ShapeMatchingConstraintCount * sizeof(Animation::CShapeMatchingConstraint);
		hipError_t cudaStatus = hipMalloc((void**)&m_dShapeMatchingConstraints, Size);
		if (cudaStatus != hipSuccess) ASSERT_TRUE(false);
		cudaStatus = hipMemcpy(m_dShapeMatchingConstraints, m_pShapeMatchingConstraint, Size, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) ASSERT_TRUE(false);
	}

	void _initDistanceParticles()
	{
		m_ParticleData[0].setInfo({ 0, 0, 0 }, m_Gravity);
		m_ParticleData[1].setInfo({ 0, 1.0, 0 }, m_Gravity);
		m_ParticleData[2].setInfo({ 1.0, 1.0, 0 }, m_Gravity);
		m_ParticleData[0].setMass(0.0);

		m_DistanceConstraints.resize(m_DistanceConstraintCount);
		m_DistanceConstraints[0].initializeConstraint(m_ParticleData, { 0, 1 }, 1.0);
		m_DistanceConstraints[1].initializeConstraint(m_ParticleData, { 1, 2 }, 1.0);

		int Size = m_DistanceConstraintCount * sizeof(Animation::CDistanceConstraint);
		hipError_t cudaStatus = hipMalloc((void**)&m_dDistanceConstraints, Size);
		if (cudaStatus != hipSuccess) ASSERT_TRUE(false);
		cudaStatus = hipMemcpy(m_dDistanceConstraints, m_DistanceConstraints.data(), Size, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) ASSERT_TRUE(false);
	}

	void _initVolumeParticles()
	{
		m_ParticleData[0].setInfo({ 0, 0, 0 }, m_Gravity);
		m_ParticleData[1].setInfo({ 0, 1.0, 0 }, m_Gravity);
		m_ParticleData[2].setInfo({ 1.0, 0, 0 }, m_Gravity);
		m_ParticleData[3].setInfo({ 0, 0, 1.0 }, m_Gravity);
		m_ParticleData[0].setMass(0.0);
		
		m_pVolumeConstraints = new Animation::CVolumeConstraint();
		std::vector<int> Indices = { 0, 1, 2, 3 };
		m_pVolumeConstraints->initializeConstraint(m_ParticleData, Indices, 1.0);

		int Size = m_VolumeConstraintCount * sizeof(Animation::CVolumeConstraint);
		hipError_t cudaStatus = hipMalloc((void**)&m_dVolumeConstraints, Size);
		if (cudaStatus != hipSuccess) ASSERT_TRUE(false);
		cudaStatus = hipMemcpy(m_dVolumeConstraints, m_pVolumeConstraints, Size, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) ASSERT_TRUE(false);
	}

	void _calAreaAndAngles(double& voArea, std::vector<double>& voAngles)
	{
		Animation::SVector3d EdgeA = m_ParticleData[1].getPosition() - m_ParticleData[0].getPosition();
		Animation::SVector3d EdgeB = m_ParticleData[2].getPosition() - m_ParticleData[0].getPosition();
		Animation::SVector3d EdgeC = m_ParticleData[2].getPosition() - m_ParticleData[1].getPosition();

		voArea = _calArea(EdgeA.norm(), EdgeB.norm(), EdgeC.norm());

		voAngles.resize(m_ParticleCount);
		EdgeA.normalize();
		EdgeB.normalize();
		EdgeC.normalize();
		voAngles[0] = EdgeA.dot(EdgeB);
		voAngles[1] = EdgeA.dot(EdgeC);
		voAngles[2] = EdgeB.dot(EdgeC);
	}

	inline double _calArea(double vEdgeA, double vEdgeB, double vEdgeC)
	{
		const double P = (vEdgeA + vEdgeB + vEdgeC) / 2;

		return sqrt(P* (P - vEdgeA) * (P - vEdgeB) * (P - vEdgeC));
	}

	void _isEqual(const Animation::SVector3d& vLeft, const Animation::SVector3d& vRight)
	{
		EXPECT_DOUBLE_EQ(vLeft.x, vRight.x);
		EXPECT_DOUBLE_EQ(vLeft.y, vRight.y);
		EXPECT_DOUBLE_EQ(vLeft.z, vRight.z);
	}

	void _isNear(const Animation::SVector3d& vLeft, const Animation::SVector3d& vRight, double vTolerance)
	{
		EXPECT_NEAR(vLeft.x, vRight.x, vTolerance);
		EXPECT_NEAR(vLeft.y, vRight.y, vTolerance);
		EXPECT_NEAR(vLeft.z, vRight.z, vTolerance);
	}

	void _initCuda()
	{
		hipError_t cudaStatus = hipSuccess;
		hipDeviceProp_t DeviceProp;
		cudaStatus = hipGetDeviceProperties(&DeviceProp, 0);
		if (cudaStatus != hipSuccess) ASSERT_TRUE(false);
		m_MaxThreadsPerBlock = DeviceProp.maxThreadsPerBlock;
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) ASSERT_TRUE(false);

		int Size = m_ParticleCount * sizeof(Animation::CParticleData);
		cudaStatus = hipMalloc((void**)&m_dParticleData, Size);
		if (cudaStatus != hipSuccess) ASSERT_TRUE(false);
		cudaStatus = hipMemcpy(m_dParticleData, m_ParticleData.data(), Size, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) ASSERT_TRUE(false);

		//std::vector<Animation::CLock> Locks(m_ParticleCount);
		//Size = m_ParticleCount * sizeof(Animation::CLock);
		//cudaStatus = hipMalloc((void**)&m_dLocks, Size);
		//if (cudaStatus != hipSuccess) ASSERT_TRUE(false);
		//cudaStatus = hipMemcpy(m_dLocks, Locks.data(), Size, hipMemcpyHostToDevice);
		//if (cudaStatus != hipSuccess) ASSERT_TRUE(false);
	}

	void _freeCuda()
	{
		hipFree(m_dDistanceConstraints);
		hipFree(m_dVolumeConstraints);
		hipFree(m_dShapeMatchingConstraints);
		//hipFree(m_dLocks);
		hipFree(m_dParticleData);
	}

	int m_ParticleCount = 0;
	std::vector<Animation::CParticleData> m_ParticleData;
	Animation::CShapeMatchingConstraint* m_pShapeMatchingConstraint = nullptr;
	std::vector<Animation::CDistanceConstraint> m_DistanceConstraints;
	Animation::CVolumeConstraint *m_pVolumeConstraints = nullptr;
	Animation::SVector3d m_Gravity = { 0, 10.0, 0 };

	int m_ShapeMatchingConstraintCount = 1;
	int m_DistanceConstraintCount = 2;
	int m_VolumeConstraintCount = 1;
	int m_MaxThreadsPerBlock = 0;
	Animation::CParticleData *m_dParticleData = nullptr;
	Animation::CLock *m_dLocks = nullptr;
	Animation::CShapeMatchingConstraint *m_dShapeMatchingConstraints = nullptr;
	Animation::CDistanceConstraint *m_dDistanceConstraints = nullptr;
	Animation::CVolumeConstraint *m_dVolumeConstraints = nullptr;
};

//**************************************
//���Ե㣺���Խ�����״ƥ��Լ���Ľ���Ƿ���ȷ
//ͨ������: 
//		1�����ӵ�λ����һ����Χ
//		2��������ɵ����/������
//		3�����Ӽ�ļнǽǶȴ������
TEST_F(CTestPositionBasedDynamics, TestShapeMatchingConstraint)
{
	m_ParticleCount = 3;
	m_ParticleData.resize(m_ParticleCount);
	_initShapeMatchingParticles();
	double ExpectArea;
	std::vector<double> ExpectAngles;
	_calAreaAndAngles(ExpectArea, ExpectAngles);

	m_ParticleData[0].setPosition({ -1.0, 0, 0 });

	_initCuda();

	Animation::solveShapeMatchingConstraints<<<1, 1>>>(m_dParticleData, m_dShapeMatchingConstraints, m_dLocks, m_ShapeMatchingConstraintCount);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) ASSERT_TRUE(false);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) ASSERT_TRUE(false);

	std::vector<Animation::CParticleData> TempParticleData(m_ParticleCount);
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(TempParticleData.data(), m_dParticleData, m_ParticleCount * sizeof(Animation::CParticleData), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) ASSERT_TRUE(false);

	m_ParticleData = TempParticleData;

	const double Epsilon = 0.5;
	_isEqual({ -1.0, 0, 0 }, m_ParticleData[0].getPosition());
	_isNear({ -1.0, 1.0, 0 }, m_ParticleData[1].getPosition(), Epsilon);
	_isNear({ 0, 0, 0 }, m_ParticleData[2].getPosition(), Epsilon);

	double ActualArea;
	std::vector<double> ActualAngles;
	_calAreaAndAngles(ActualArea, ActualAngles);

	EXPECT_DOUBLE_EQ(ExpectArea, ActualArea);
	ASSERT_EQ(ExpectAngles.size(), ActualAngles.size());
	for (int i = 0; i < ExpectAngles.size(); i++)
	{
		EXPECT_NEAR(ExpectAngles[i], ActualAngles[i], 1e-6);
	}
}

//**************************************
//���Ե㣺���Խ������Լ���Ľ���Ƿ���ȷ
//ͨ������: 
//		1�����ӵ�λ����һ����Χ
//		2�����Ӽ�ľ���仯����
TEST_F(CTestPositionBasedDynamics, TestDistanceConstraint)
{
	m_ParticleCount = 3;
	m_ParticleData.resize(m_ParticleCount);
	_initDistanceParticles();
	double ExpectDis0 = (m_ParticleData[0].getPosition() - m_ParticleData[1].getPosition()).norm();
	double ExpectDis1 = (m_ParticleData[1].getPosition() - m_ParticleData[2].getPosition()).norm();

	m_ParticleData[0].setPosition({ -1.0, 0, 0 });

	_initCuda();

	for (int i = 0; i < 20; i++)
	{
		Animation::solveDistanceConstraints<<<1, 2>>>(m_dParticleData, m_dDistanceConstraints, m_dLocks, m_DistanceConstraintCount);
		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) ASSERT_TRUE(false);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) ASSERT_TRUE(false);
	}

	std::vector<Animation::CParticleData> TempParticleData(m_ParticleCount);
	// Copy output vector from GPU buffer to host memory.
	hipError_t cudaStatus = hipMemcpy(TempParticleData.data(), m_dParticleData, m_ParticleCount * sizeof(Animation::CParticleData), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) ASSERT_TRUE(false);

	m_ParticleData = TempParticleData;

	double Dis0 = (m_ParticleData[0].getPosition() - m_ParticleData[1].getPosition()).norm();
	double Dis1 = (m_ParticleData[1].getPosition() - m_ParticleData[2].getPosition()).norm();
	EXPECT_NEAR(ExpectDis0, Dis0, 1e-4);
	EXPECT_NEAR(ExpectDis1, Dis1, 1e-4);
}

//**************************************
//���Ե㣺���Խ������Լ���Ľ���Ƿ���ȷ
//ͨ������: 
//		1�����ӵ�λ����һ����Χ
//		2�����Ӽ������仯����
TEST_F(CTestPositionBasedDynamics, TestVolumeConstraint)
{
	m_ParticleCount = 4;
	m_ParticleData.resize(m_ParticleCount);
	_initVolumeParticles();
	double ExpectVolume = 0.5 / 3;

	m_ParticleData[0].setPosition({ -1.0, 0, 0 });

	_initCuda();

	for (int i = 0; i < 3; i++)
	{
		Animation::solveVolumeConstraints<<<1, 1>>>(m_dParticleData, m_dVolumeConstraints, m_dLocks, m_VolumeConstraintCount);
		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) ASSERT_TRUE(false);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) ASSERT_TRUE(false);
	}

	std::vector<Animation::CParticleData> TempParticleData(m_ParticleCount);
	// Copy output vector from GPU buffer to host memory.
	hipError_t cudaStatus = hipMemcpy(TempParticleData.data(), m_dParticleData, m_ParticleCount * sizeof(Animation::CParticleData), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) ASSERT_TRUE(false);

	m_ParticleData = TempParticleData;

	double Volume = fabs(1.0 / 6.0 * (m_ParticleData[3].getPosition() - m_ParticleData[0].getPosition()).dot((m_ParticleData[2].getPosition() - m_ParticleData[0].getPosition()).cross(m_ParticleData[1].getPosition() - m_ParticleData[0].getPosition())));
	EXPECT_NEAR(ExpectVolume, Volume, 0.05);
}