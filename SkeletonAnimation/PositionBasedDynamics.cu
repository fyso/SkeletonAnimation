#include "hip/hip_runtime.h"
#pragma once
#include "DistanceConstraint.h"
#include "VolumeConstraint.h"
#include "ShapeMatchingConstraint.h"
#include "ProjectionConstraint.h"
#include <stdlib.h>
#include <stdio.h>
#include "Solvers.cu"

namespace Animation
{
	__global__ void updateBoneParticles(CParticleData* vioParticleData, int* vIndices, SVector3d* vPos, int vSize)
	{
		int Index = blockIdx.x * blockDim.x + threadIdx.x;
		if (Index >= vSize) return;

		vioParticleData[vIndices[Index]].setPosition(vPos[Index]);
	}

	__global__ void predictPosition(CParticleData* vioParticleData, double vTimeStep, int vSize)
	{
		int Index = blockIdx.x * blockDim.x + threadIdx.x;
		if (Index >= vSize) return;

		vioParticleData[Index].fetchLastPosition() = vioParticleData[Index].getOldPosition();
		vioParticleData[Index].fetchOldPosition() = vioParticleData[Index].getPosition();

		if (vioParticleData[Index].getMass() > DBL_EPSILON)
		{
			vioParticleData[Index].fetchVelocity() += vioParticleData[Index].getAcceleration() * vTimeStep;
			vioParticleData[Index].fetchPosition() += vioParticleData[Index].getVelocity() * vTimeStep;
		}
	}

	__global__ void solveDistanceConstraints(CParticleData* vioParticleData, CDistanceConstraint* vDistanceConstraints, int vSize, int vOffset)
	{
		int Index = vOffset + blockIdx.x * blockDim.x + threadIdx.x;
		if (Index >= vSize) return;

		CDistanceConstraint DistanceConstraint = vDistanceConstraints[Index];
		int* ParticleIndices = DistanceConstraint.getParticleIndices();
		unsigned int Index0 = ParticleIndices[0];
		unsigned int Index1 = ParticleIndices[1];

		SVector3d X0 = vioParticleData[Index0].getPosition();
		SVector3d X1 = vioParticleData[Index1].getPosition();
		double InvMass0 = vioParticleData[Index0].getInverseMass();
		double InvMass1 = vioParticleData[Index1].getInverseMass();

		double RestDistance = DistanceConstraint.getRestDistance();
		double Stiffness = DistanceConstraint.getStiffness();
		SVector3d Correct0, Correct1;
		bool Result = solveDistanceConstraint(X0, InvMass0, X1, InvMass1, RestDistance, Stiffness, Correct0, Correct1);
		if (Result)
		{
			if (InvMass0 > DBL_EPSILON)
				vioParticleData[Index0].fetchPosition() += Correct0;
			if (InvMass1 > DBL_EPSILON)
				vioParticleData[Index1].fetchPosition() += Correct1;
		}
	}

	__global__ void solveVolumeConstraints(CParticleData* vioParticleData, CVolumeConstraint* vVolumeConstraints, int vSize, int vOffset)
	{
		int Index = vOffset + blockIdx.x * blockDim.x + threadIdx.x;
		if (Index >= vSize) return;

		CVolumeConstraint VolumeConstraint = vVolumeConstraints[Index];
		int* ParticleIndices = VolumeConstraint.getParticleIndices();
		unsigned int Index0 = ParticleIndices[0];
		unsigned int Index1 = ParticleIndices[1];
		unsigned int Index2 = ParticleIndices[2];
		unsigned int Index3 = ParticleIndices[3];

		SVector3d X0 = vioParticleData[Index0].getPosition();
		SVector3d X1 = vioParticleData[Index1].getPosition();
		SVector3d X2 = vioParticleData[Index2].getPosition();
		SVector3d X3 = vioParticleData[Index3].getPosition();

		double InvMass0 = vioParticleData[Index0].getInverseMass();
		double InvMass1 = vioParticleData[Index1].getInverseMass();
		double InvMass2 = vioParticleData[Index2].getInverseMass();
		double InvMass3 = vioParticleData[Index3].getInverseMass();

		double RestVolume = VolumeConstraint.getRestVolume();
		double Stiffness = VolumeConstraint.getStiffness();
		SVector3d Correct0, Correct1, Correct2, Correct3;
		bool Result = solveVolumeConstraint(X0, InvMass0, X1, InvMass1, X2, InvMass2, X3, InvMass3, RestVolume, Stiffness, Stiffness, Correct0, Correct1, Correct2, Correct3);
		if (Result)
		{
			if (InvMass0 > DBL_EPSILON)
				vioParticleData[Index0].fetchPosition() += Correct0;
			if (InvMass1 > DBL_EPSILON)
				vioParticleData[Index1].fetchPosition() += Correct1;
			if (InvMass2 > DBL_EPSILON)
				vioParticleData[Index2].fetchPosition() += Correct2;
			if (InvMass3 > DBL_EPSILON)
				vioParticleData[Index3].fetchPosition() += Correct3;
		}
	}

	__global__ void solveShapeMatchingConstraints(CParticleData* vioParticleData, CShapeMatchingConstraint* vShapeMatchingConstraints, int* vParticleRegionCount, int vSize, int vOffset)
	{
		int Index = vOffset + blockIdx.x * blockDim.x + threadIdx.x;
		if (Index >= vSize) return;

		CShapeMatchingConstraint ShapeMatchingConstraint = vShapeMatchingConstraints[Index];
		int ParticleCount = ShapeMatchingConstraint.getParticleCount();
		int* ParticleIndices = ShapeMatchingConstraint.getParticleIndices();
		SVector3d CurX[27];
		for (int i = 0; i < ParticleCount; i++)
		{
			CurX[i] = vioParticleData[ParticleIndices[i]].getPosition();
		}

		SVector3d* X0 = ShapeMatchingConstraint.getPosition0();
		double* Weight = ShapeMatchingConstraint.getWeight();
		SVector3d RestCm = ShapeMatchingConstraint.getRestCm();
		SMatrix3d InvRestMat = ShapeMatchingConstraint.getInvRestMat();
		double* Stiffness = ShapeMatchingConstraint.getStiffness();
		SVector3d Correct[27];
		bool Result = solveShapeMathingConstrint(X0, CurX, Weight, ParticleCount, RestCm, InvRestMat, Stiffness, Correct);
		if (Result)
		{
			for (unsigned int i = 0; i < ParticleCount; i++)
			{
				// Important: Divide position correction by the number of clusters which contain the vertex. 
				if (Weight[i] > DBL_EPSILON)
				{
					const int ParticleIndex = ParticleIndices[i];
					int ClusterCount = vParticleRegionCount[ParticleIndex];
					SVector3d DeltaX = Correct[i] * (1.0 / ClusterCount);
					vioParticleData[ParticleIndex].fetchPosition() += DeltaX;
				}
			}
		}
	}

	__global__ void projectionCostraints(CParticleData* vioParticleData, CProjectionConstraint* vProjectionConstraints, int vSize)
	{
		int Index = blockIdx.x * blockDim.x + threadIdx.x;
		if (Index >= vSize) return;

		CProjectionConstraint ProjectionConstraint = vProjectionConstraints[Index];
		SVector3d Position;
		const int Count = ProjectionConstraint.getParticleCount();
		int* ParticleIndices = ProjectionConstraint.getParticleIndices();
		double SumMass = 0.0;
		for (int i = 0; i < Count; i++)
		{
			const int ParticleIndex = ParticleIndices[i];
			double Mass = vioParticleData[ParticleIndex].getMass();
			Position += vioParticleData[ParticleIndex].getPosition() * Mass;
			SumMass += Mass;
		}
		if (SumMass > DBL_EPSILON)
		{
			Position = Position / SumMass;
			for (int i = 0; i < Count; i++)
			{
				const int ParticleIndex = ParticleIndices[i];
				vioParticleData[ParticleIndex].setPosition(Position);
			}
		}
	}

	__global__ void updateParticlePosition(CParticleData* vioParticleData, SVector3d* voParticlePosition, int vSize)
	{
		int Index = blockIdx.x * blockDim.x + threadIdx.x;
		if (Index >= vSize) return;

		voParticlePosition[Index] = vioParticleData[Index].getPosition();
	}

	__global__ void updateVelocity(CParticleData* vioParticleData, double vTimeStep, double vDampFactor, int vSize)
	{
		int Index = blockIdx.x * blockDim.x + threadIdx.x;
		if (Index >= vSize) return;

		if (vioParticleData[Index].getMass() > DBL_EPSILON)
		{
			vioParticleData[Index].fetchVelocity() = (vioParticleData[Index].getPosition() - vioParticleData[Index].getOldPosition()) * (1.0 / vTimeStep);
			vioParticleData[Index].fetchVelocity() = vioParticleData[Index].getVelocity() * (1.0 - vDampFactor);
		}
	}

	__global__ void collide(CParticleData* vioParticleData, int vSize)
	{
		int Index = blockIdx.x * blockDim.x + threadIdx.x;
		if (Index >= vSize) return;

		if (vioParticleData[Index].getPosition().y < 0.0)
		{
			vioParticleData[Index].fetchPosition().y = 0.0;
			vioParticleData[Index].fetchVelocity().y *= -1.0;
		}
	}
}