#include "hip/hip_runtime.h"
#pragma once
#include "CommonMath.h"

namespace Animation
{
	__device__ bool solveDistanceConstraint(const SVector3d &vPos0, double vInvMass0, const SVector3d &vPos1, double vInvMass1,
		const double vRestLength, const double vStiffness, SVector3d &voCorrect0, SVector3d &voCorrect1)
	{
		double WeightSum = vInvMass0 + vInvMass1;
		if (WeightSum < DBL_EPSILON)
			return false;

		SVector3d Vector = vPos1 - vPos0;
		double Distance = Vector.norm();
		Vector.normalize();

		SVector3d Correct = Vector * vStiffness * (Distance - vRestLength) / WeightSum;

		voCorrect0 = Correct * vInvMass0;
		voCorrect1 = Correct * -1.0 * vInvMass1;
		return true;
	}

	__device__ bool solveVolumeConstraint(const SVector3d &vPos0, double vInvMass0, const SVector3d &vPos1, double vInvMass1,
		const SVector3d &vPos2, double vInvMass2, const SVector3d &vPos3, double vInvMass3,
		const double vRestVolume, const double vNegVolumeStiffness, const double vPosVolumeStiffness,
		SVector3d &voCorrect0, SVector3d &voCorrect1, SVector3d &voCorrect2, SVector3d &voCorrect3)
	{
		double Volume = 1.0 / 6.0 * (vPos1 - vPos0).cross(vPos2 - vPos0).dot(vPos3 - vPos0);

		voCorrect0.setZero(); voCorrect1.setZero(); voCorrect2.setZero(); voCorrect3.setZero();

		if (vPosVolumeStiffness == 0.0 && Volume > 0.0)
			return false;

		if (vNegVolumeStiffness == 0.0 && Volume < 0.0)
			return false;

		SVector3d Grad0 = (vPos1 - vPos2).cross(vPos3 - vPos2);
		SVector3d Grad1 = (vPos2 - vPos0).cross(vPos3 - vPos0);
		SVector3d Grad2 = (vPos0 - vPos1).cross(vPos3 - vPos1);
		SVector3d Grad3 = (vPos1 - vPos0).cross(vPos2 - vPos0);

		double Lambda =
			vInvMass0 * Grad0.squaredNorm() +
			vInvMass1 * Grad1.squaredNorm() +
			vInvMass2 * Grad2.squaredNorm() +
			vInvMass3 * Grad3.squaredNorm();

		if (fabs(Lambda) < DBL_EPSILON)
			return false;

		if (Volume < 0.0)
			Lambda = vNegVolumeStiffness * (Volume - vRestVolume) / Lambda;
		else
			Lambda = vPosVolumeStiffness * (Volume - vRestVolume) / Lambda;

		voCorrect0 = Grad0 * -1.0 * Lambda * vInvMass0;
		voCorrect1 = Grad1 * -1.0 * Lambda * vInvMass1;
		voCorrect2 = Grad2 * -1.0 * Lambda * vInvMass2;
		voCorrect3 = Grad3 * -1.0 * Lambda * vInvMass3;

		return true;
	}

	__device__ bool solveShapeMathingConstrint(
		const SVector3d vX0[], const SVector3d vX[], const double vInvMasses[], const int vPointCount,
		const SVector3d &vRestCm,
		const SMatrix3d &vInvRestMat,
		const double vStiffness[],
		SVector3d voCorrect[])
	{
		for (int i = 0; i < vPointCount; i++)
			voCorrect[i].setZero();

		// center of mass
		SVector3d Cm(0.0);
		double WeightSum = 0.0;
		for (int i = 0; i < vPointCount; i++)
		{
			double Weight = 1.0 / (vInvMasses[i] + DBL_EPSILON);
			Cm += vX[i] * Weight;
			WeightSum += Weight;
		}
		if (WeightSum == 0.0)
			return false;
		Cm = Cm / WeightSum;

		SMatrix3d Matrix;
		Matrix.setZero();
		for (int i = 0; i < vPointCount; i++) {
			SVector3d Q = vX0[i] - vRestCm;
			SVector3d P = vX[i] - Cm;

			double Weight = 1.0 / (vInvMasses[i] + DBL_EPSILON);
			P = P * Weight;

			Matrix(0, 0) += P[0] * Q[0]; Matrix(0, 1) += P[0] * Q[1]; Matrix(0, 2) += P[0] * Q[2];
			Matrix(1, 0) += P[1] * Q[0]; Matrix(1, 1) += P[1] * Q[1]; Matrix(1, 2) += P[1] * Q[2];
			Matrix(2, 0) += P[2] * Q[0]; Matrix(2, 1) += P[2] * Q[1]; Matrix(2, 2) += P[2] * Q[2];
		}

		Matrix = Matrix * vInvRestMat;// ������ת�����ı任����mat

		SMatrix3d R = Matrix;
		polarDecompositionStable(Matrix, 1e-6, R);

		for (int i = 0; i < vPointCount; i++) {
			SVector3d Goal = Cm + R * (vX0[i] - vRestCm);
			voCorrect[i] = (Goal - vX[i]) * vStiffness[i];
		}

		return true;
	}
}