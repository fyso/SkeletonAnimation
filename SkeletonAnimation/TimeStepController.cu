#include "hip/hip_runtime.h"
#pragma once
#include "TimeStepController.h"
#include "PositionBasedDynamics.cu"
#include <iostream>

int *dBoneIndices = nullptr;
Animation::SVector3d *dBonePos = nullptr;
Animation::SVector3d *dParticlePos = nullptr;

Animation::CTimeStepController::CTimeStepController(int vIterationCount, double vTimeStep, double vDampFactor)
	: m_IterationCount(vIterationCount), m_TimeStep(vTimeStep), m_DampFactor(vDampFactor)
{
}

Animation::CTimeStepController::~CTimeStepController()
{
	__freeCudaMemory();
}

//*********************************************************************
//FUNCTION:
void Animation::CTimeStepController::init(CSimulationModel& vSimulationModel, int vOriginParticleCount)
{
	m_OriginParticleCount = vOriginParticleCount;

	hipDeviceProp_t DeviceProp;
	HANDLE_ERROR(hipGetDeviceProperties(&DeviceProp, 0));
	m_MaxThreadsPerBlock = DeviceProp.maxThreadsPerBlock / 4;
	HANDLE_ERROR(hipSetDevice(0));

	std::vector<CParticleData> ParticleData;
	vSimulationModel.dumpSimulationParticle(ParticleData);
	m_ParticleCount = ParticleData.size();

	int Size = m_ParticleCount * sizeof(CParticleData);
	HANDLE_ERROR(hipMalloc((void**)&m_dParticleData, Size));
	HANDLE_ERROR(hipMemcpy(m_dParticleData, ParticleData.data(), Size, hipMemcpyHostToDevice));

	Size = m_OriginParticleCount * sizeof(SVector3d);
	HANDLE_ERROR(hipMalloc((void**)&dParticlePos, Size));

	std::vector<CDistanceConstraint> DistanceConstraints;
	vSimulationModel.dumpDistanceConstraints(DistanceConstraints);
	m_DistanceConstraintCount = DistanceConstraints.size();
	if (m_DistanceConstraintCount > 0)
	{
		m_DistanceConstraintOffset.push_back(0);
		std::vector<int> DistanceConstraintsGroupTemp = vSimulationModel.getDistanceConstraintsGroupOffset();
		m_DistanceConstraintOffset.insert(m_DistanceConstraintOffset.end(), DistanceConstraintsGroupTemp.begin(), DistanceConstraintsGroupTemp.end());
		Size = m_DistanceConstraintCount * sizeof(CDistanceConstraint);
		HANDLE_ERROR(hipMalloc((void**)&m_dDistanceConstraints, Size));
		HANDLE_ERROR(hipMemcpy(m_dDistanceConstraints, DistanceConstraints.data(), Size, hipMemcpyHostToDevice));
	}
	
	std::vector<CProjectionConstraint> DistanceProjectionConstraints;
	vSimulationModel.dumpDistanceProjectionConstraints(DistanceProjectionConstraints);
	m_DistanceProjectionConstraintCount = DistanceProjectionConstraints.size();
	if (m_DistanceProjectionConstraintCount > 0)
	{
		Size = m_DistanceProjectionConstraintCount * sizeof(CProjectionConstraint);
		HANDLE_ERROR(hipMalloc((void**)&m_dDistanceProjectionConstraints, Size));
		HANDLE_ERROR(hipMemcpy(m_dDistanceProjectionConstraints, DistanceProjectionConstraints.data(), Size, hipMemcpyHostToDevice));
	}	

	std::vector<CVolumeConstraint> VolumeConstraints;
	vSimulationModel.dumpVolumeConstraints(VolumeConstraints);
	m_VolumeConstraintCount = VolumeConstraints.size();
	if (m_VolumeConstraintCount > 0)
	{
		m_VolumeConstraintOffset.push_back(0);
		std::vector<int> VolumeConstraintsGroupTemp = vSimulationModel.getVolumeConstraintsGroupOffset();
		m_VolumeConstraintOffset.insert(m_VolumeConstraintOffset.end(), VolumeConstraintsGroupTemp.begin(), VolumeConstraintsGroupTemp.end());
		Size = m_VolumeConstraintCount * sizeof(CVolumeConstraint);
		HANDLE_ERROR(hipMalloc((void**)&m_dVolumeConstraints, Size));
		HANDLE_ERROR(hipMemcpy(m_dVolumeConstraints, VolumeConstraints.data(), Size, hipMemcpyHostToDevice));
	}
	
	std::vector<CProjectionConstraint> VolumeProjectionConstraints;
	vSimulationModel.dumpVolumeProjectionConstraints(VolumeProjectionConstraints);
	m_VolumeProjectionConstraintCount = VolumeProjectionConstraints.size();
	if (m_VolumeProjectionConstraintCount > 0)
	{
		Size = m_VolumeProjectionConstraintCount * sizeof(CProjectionConstraint);
		HANDLE_ERROR(hipMalloc((void**)&m_dVolumeProjectionConstraints, Size));
		HANDLE_ERROR(hipMemcpy(m_dVolumeProjectionConstraints, VolumeProjectionConstraints.data(), Size, hipMemcpyHostToDevice));
	}
	
	std::vector<CShapeMatchingConstraint> ShapeMatchingConstraints;
	vSimulationModel.dumpShapeMatchingConstraints(ShapeMatchingConstraints);
	m_ShapeMatchingConstraintCount = ShapeMatchingConstraints.size();
	std::vector<int> particleRegionCount = vSimulationModel.getParticleRelatedClusterCount();
	if (m_ShapeMatchingConstraintCount > 0)
	{
		m_ShapeMatchingConstraintOffset.push_back(0);
		std::vector<int> ShapeMatchingConstraintsGroupTemp = vSimulationModel.getShapeMatchingConstraintsGroupOffset();
		m_ShapeMatchingConstraintOffset.insert(m_ShapeMatchingConstraintOffset.end(), ShapeMatchingConstraintsGroupTemp.begin(), ShapeMatchingConstraintsGroupTemp.end());
		Size = m_ShapeMatchingConstraintCount * sizeof(CShapeMatchingConstraint);
		HANDLE_ERROR(hipMalloc((void**)&m_dShapeMatchingConstraints, Size));
		HANDLE_ERROR(hipMemcpy(m_dShapeMatchingConstraints, ShapeMatchingConstraints.data(), Size, hipMemcpyHostToDevice));

		Size = particleRegionCount.size() * sizeof(int);
		HANDLE_ERROR(hipMalloc((void**)&m_dParticleRegionCount, Size));
		HANDLE_ERROR(hipMemcpy(m_dParticleRegionCount, particleRegionCount.data(), Size, hipMemcpyHostToDevice));
	}
	
	std::vector<CProjectionConstraint> ShapeMatchingProjectionConstraints;
	vSimulationModel.dumpShapeMatchingProjectionConstraints(ShapeMatchingProjectionConstraints);
	m_ShapeMatchingProjectionConstraintCount = ShapeMatchingProjectionConstraints.size();
	if (m_ShapeMatchingProjectionConstraintCount > 0)
	{
		Size = m_ShapeMatchingProjectionConstraintCount * sizeof(CProjectionConstraint);
		HANDLE_ERROR(hipMalloc((void**)&m_dShapeMatchingProjectionConstraints, Size));
		HANDLE_ERROR(hipMemcpy(m_dShapeMatchingProjectionConstraints, ShapeMatchingProjectionConstraints.data(), Size, hipMemcpyHostToDevice));
	}	
}

//*********************************************************************
//FUNCTION:
__host__ bool Animation::CTimeStepController::step()
{
	int BlockSize = std::min(m_MaxThreadsPerBlock, m_ParticleCount);
	int GridSize = (m_ParticleCount + BlockSize - 1) / BlockSize;
	predictPosition<<<GridSize, BlockSize>>>(m_dParticleData, m_TimeStep, m_ParticleCount);
	HANDLE_ERROR(hipGetLastError());
	HANDLE_ERROR(hipDeviceSynchronize());

	for (int i = 0; i < m_IterationCount; i++)
	{
		for (int k = 0; k < m_DistanceConstraintOffset.size()-1; k++)
		{
			int GroupSize = m_DistanceConstraintOffset[k + 1] - m_DistanceConstraintOffset[k];
			BlockSize = std::min(m_MaxThreadsPerBlock, GroupSize);
			GridSize = (GroupSize + BlockSize -1) / BlockSize;
			solveDistanceConstraints<<<GridSize, BlockSize>>>(m_dParticleData, m_dDistanceConstraints, m_DistanceConstraintOffset[k + 1], m_DistanceConstraintOffset[k]);
			HANDLE_ERROR(hipGetLastError());
		}

		if (m_DistanceProjectionConstraintCount > 0)
		{
			BlockSize = std::min(m_MaxThreadsPerBlock, m_DistanceProjectionConstraintCount);
			GridSize = (m_DistanceProjectionConstraintCount + BlockSize - 1) / BlockSize;
			projectionCostraints <<<GridSize, BlockSize >>> (m_dParticleData, m_dDistanceProjectionConstraints, m_DistanceProjectionConstraintCount);
			HANDLE_ERROR(hipGetLastError());
			//HANDLE_ERROR(hipDeviceSynchronize());
		}		

		for (int k = 0; k < m_VolumeConstraintOffset.size()-1; k++)
		{
			int GroupSize = m_VolumeConstraintOffset[k + 1] - m_VolumeConstraintOffset[k];
			BlockSize = std::min(m_MaxThreadsPerBlock / 2, GroupSize);
			GridSize = (GroupSize + BlockSize - 1) / BlockSize;
			solveVolumeConstraints<<<GridSize, BlockSize>>>(m_dParticleData, m_dVolumeConstraints, m_VolumeConstraintOffset[k + 1], m_VolumeConstraintOffset[k]);
			HANDLE_ERROR(hipGetLastError());
		}

		if (m_VolumeProjectionConstraintCount > 0)
		{
			BlockSize = std::min(m_MaxThreadsPerBlock / 2, m_VolumeProjectionConstraintCount);
			GridSize = (m_VolumeProjectionConstraintCount + BlockSize - 1) / BlockSize;
			projectionCostraints <<<GridSize, BlockSize >>> (m_dParticleData, m_dVolumeProjectionConstraints, m_VolumeProjectionConstraintCount);
			HANDLE_ERROR(hipGetLastError());
			//HANDLE_ERROR(hipDeviceSynchronize());
		}

		for (int k = 0; k < m_ShapeMatchingConstraintOffset.size()-1; k++)
		{
			int GroupSize = m_ShapeMatchingConstraintOffset[k + 1] - m_ShapeMatchingConstraintOffset[k];
			BlockSize = std::min(m_MaxThreadsPerBlock / 2, GroupSize);
			GridSize = (GroupSize + BlockSize - 1) / BlockSize;
			solveShapeMatchingConstraints<<<GridSize, BlockSize >>>(m_dParticleData, m_dShapeMatchingConstraints, m_dParticleRegionCount, m_ShapeMatchingConstraintOffset[k + 1], m_ShapeMatchingConstraintOffset[k]);
			HANDLE_ERROR(hipGetLastError());
		}

		if (m_ShapeMatchingProjectionConstraintCount > 0)
		{
			BlockSize = std::min(m_MaxThreadsPerBlock / 2, m_ShapeMatchingProjectionConstraintCount);
			GridSize = (m_ShapeMatchingProjectionConstraintCount + BlockSize - 1) / BlockSize;
			projectionCostraints <<<GridSize, BlockSize >>> (m_dParticleData, m_dShapeMatchingProjectionConstraints, m_ShapeMatchingProjectionConstraintCount);
			HANDLE_ERROR(hipGetLastError());
			HANDLE_ERROR(hipDeviceSynchronize());
		}
	}

	BlockSize = std::min(m_MaxThreadsPerBlock / 2, m_OriginParticleCount);
	GridSize = (m_OriginParticleCount + BlockSize - 1) / BlockSize;
	updateParticlePosition<<<GridSize, BlockSize>>>(m_dParticleData, dParticlePos, m_OriginParticleCount);
	HANDLE_ERROR(hipGetLastError());

	BlockSize = std::min(m_MaxThreadsPerBlock / 2, m_ParticleCount);
	GridSize = (m_ParticleCount + BlockSize - 1) / BlockSize;
	updateVelocity<<<GridSize, BlockSize>>>(m_dParticleData, m_TimeStep, m_DampFactor, m_ParticleCount);
	HANDLE_ERROR(hipGetLastError());
	HANDLE_ERROR(hipDeviceSynchronize());

	return true;
}

//*********************************************************************
//FUNCTION:
void Animation::CTimeStepController::updateBoneParticlesWithCuda(const std::vector<int>& vIndices, const std::vector<SVector3d>& vPos)
{
	const int BoneParticleCount = vPos.size();
	const int Size = BoneParticleCount * sizeof(SVector3d);
	const int Indice_Size = BoneParticleCount * sizeof(int);
	_ASSERTE(BoneParticleCount == vIndices.size());
	if (dBoneIndices == nullptr)
	{
		HANDLE_ERROR(hipMalloc((void**)&dBoneIndices, Indice_Size));
	}
	if (dBonePos == nullptr)
	{
		HANDLE_ERROR(hipMalloc((void**)&dBonePos, Size));
	}

	HANDLE_ERROR(hipMemcpy(dBoneIndices, vIndices.data(), Indice_Size, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dBonePos, vPos.data(), Size, hipMemcpyHostToDevice));

	int BlockSize = std::min(m_MaxThreadsPerBlock, BoneParticleCount);
	int GridSize = (BoneParticleCount + BlockSize - 1) / BlockSize;
	updateBoneParticles<<<GridSize, BlockSize>>>(m_dParticleData, dBoneIndices, dBonePos, BoneParticleCount);
	HANDLE_ERROR(hipGetLastError());
	HANDLE_ERROR(hipDeviceSynchronize());
}

//*********************************************************************
//FUNCTION:
std::vector<Eigen::Vector3d> Animation::CTimeStepController::getPositions()
{
	std::vector<Eigen::Vector3d> Positions;

	//HANDLE_ERROR(hipDeviceSynchronize());
	std::vector<SVector3d> TempParticlePosition(m_OriginParticleCount);
	HANDLE_ERROR(hipMemcpy(TempParticlePosition.data(), dParticlePos, m_OriginParticleCount * sizeof(SVector3d), hipMemcpyDeviceToHost));

	Positions.resize(m_OriginParticleCount);
	for (int i = 0; i < m_OriginParticleCount; i++)
	{
		Eigen::Vector3d Position(TempParticlePosition[i].x, TempParticlePosition[i].y, TempParticlePosition[i].z);
		Positions[i] = Position;
	}

	return Positions;
}

//*********************************************************************
//FUNCTION:
void Animation::CTimeStepController::resetParticleData(CSimulationModel& vSimulationModel)
{
	if (m_dParticleData == nullptr)
		return;
	std::vector<CParticleData> ParticleData;
	vSimulationModel.dumpSimulationParticle(ParticleData);
	int Size = m_ParticleCount * sizeof(CParticleData);
	HANDLE_ERROR(hipMemcpy(m_dParticleData, ParticleData.data(), Size, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipDeviceSynchronize());
}

//*********************************************************************
//FUNCTION:
void Animation::CTimeStepController::__freeCudaMemory()
{
	if (nullptr != m_dShapeMatchingConstraints)
		hipFree(m_dShapeMatchingConstraints);
	
	if(nullptr != m_dShapeMatchingProjectionConstraints)
		hipFree(m_dShapeMatchingProjectionConstraints);
	
	if(nullptr != m_dVolumeConstraints)
		hipFree(m_dVolumeConstraints);

	if(nullptr != m_dVolumeProjectionConstraints)
		hipFree(m_dVolumeProjectionConstraints);

	if(nullptr != m_dDistanceConstraints)
		hipFree(m_dDistanceConstraints);
	
	if(nullptr != m_dDistanceProjectionConstraints)
		hipFree(m_dDistanceProjectionConstraints);
	
	if(nullptr != m_dParticleData)
		hipFree(m_dParticleData);
	
	if(nullptr != dBoneIndices)
		hipFree(dBoneIndices);
	
	if(nullptr != dBonePos)
		hipFree(dBonePos);
	
	if(nullptr != dParticlePos)
		hipFree(dParticlePos);

	if (nullptr != m_dParticleRegionCount)
		hipFree(m_dParticleRegionCount);
}